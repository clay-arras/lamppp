#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <cuda/std/array>
#include <cuda/std/tuple>
#include "lamppp/tensor/cuda/expand.cuh"
#include "lamppp/tensor/cuda/kernels.cuh"
#include "lamppp/tensor/cuda/list_ptr.cuh"

namespace lmp::tensor::detail::cuda {

template <typename PtrList, typename OpFn>
__global__ void vectorized_expand_kernel(PtrList ptr_, OpFn fn_, size_t size,
                                         const CUDAOffsetUtil<kNArgs>* align) {
  for (size_t i = (blockIdx.x * blockDim.x) + threadIdx.x; i < size;
       i += gridDim.x * blockDim.x) {  // grid stride loop trick
    ::cuda::std::array offsets = align->get(i);
    ptr_.set_Out(i,
                 fn_(::cuda::std::get<1>(ptr_.fns)(ptr_.data[1], offsets[1]),
                     ::cuda::std::get<2>(ptr_.fns)(ptr_.data[2], offsets[2])));
  }
}

template <typename PtrList, typename OpFn>
void expand_kernel_launcher(PtrList ptr_, OpFn fn_, size_t size,
                            const CUDAOffsetUtil<kNArgs>* align) {
  size_t threads = 256;
  size_t blocks = std::min((size + threads - 1) / threads, 1024UL);
  ListDevicePtr<CUDAOffsetUtil<kNArgs>> d_align(align, 1);
  vectorized_expand_kernel<<<blocks, threads>>>(ptr_, fn_, size, d_align.get());

  LMP_CUDA_INTERNAL_ASSERT(hipDeviceSynchronize())
      << "expand_kernel_launcher: kernel failed.";
}

template <template <typename> class OpFunctor, typename... Args>
void expand_dispatch_handler(BinaryMetaHandler& meta, Args&&... args) {
  LMP_DISPATCH_ALL_TYPES(meta.out().type(), [&] {
    using out_dtype_t = scalar_t;
    LMP_DISPATCH_ALL_TYPES(meta.in()[0]->type(), [&] {
      using arg1_dtype_t = scalar_t;
      LMP_DISPATCH_ALL_TYPES(meta.in()[1]->type(), [&] {
        using arg2_dtype_t = scalar_t;
        expand_kernel_launcher(
            internal::CUDAPtrPack<out_dtype_t, arg1_dtype_t, arg2_dtype_t>(
                static_cast<out_dtype_t*>(meta.out().data()),
                static_cast<arg1_dtype_t*>(meta.in()[0]->data()),
                static_cast<arg2_dtype_t*>(meta.in()[1]->data())),
            OpFunctor<out_dtype_t>(std::forward<Args>(args)...),
            meta.out().numel(),
            static_cast<const CUDAOffsetUtil<kNArgs>*>(meta.offset()));
      });
    });
  });
}

template void expand_dispatch_handler<AddFunctor>(BinaryMetaHandler&);
template void expand_dispatch_handler<SubFunctor>(BinaryMetaHandler&);
template void expand_dispatch_handler<MulFunctor>(BinaryMetaHandler&);
template void expand_dispatch_handler<DivFunctor>(BinaryMetaHandler&);
template void expand_dispatch_handler<PowFunctor>(BinaryMetaHandler&);
template void expand_dispatch_handler<EqFunctor>(BinaryMetaHandler&);
template void expand_dispatch_handler<NeFunctor>(BinaryMetaHandler&);
template void expand_dispatch_handler<GeFunctor>(BinaryMetaHandler&);
template void expand_dispatch_handler<GtFunctor>(BinaryMetaHandler&);
template void expand_dispatch_handler<LeFunctor>(BinaryMetaHandler&);
template void expand_dispatch_handler<LtFunctor>(BinaryMetaHandler&);

}  // namespace lmp::tensor::detail::cuda