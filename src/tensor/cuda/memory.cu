#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <thrust/device_ptr.h>
#include <cstdint>
#include <cuda/std/array>
#include "lamppp/common/assert.hpp"
#include "lamppp/common/macros.hpp"
#include "lamppp/tensor/cpu/memory.hpp"
#include "lamppp/tensor/cuda/list_ptr.cuh"
#include "lamppp/tensor/cuda/memory.cuh"
#include "lamppp/tensor/dispatch_type.hpp"
#include "lamppp/tensor/native/memory_ops.hpp"

namespace lmp::tensor::detail::cuda {

DataPtr empty_cuda(size_t byte_size) {
  void* raw = nullptr;
  LMP_CUDA_CHECK(hipMallocAsync(&raw, byte_size, nullptr))
      << "empty_cuda: hipMalloc failed.";
  return DataPtr(raw, [byte_size](void* ptr) {
    LMP_CUDA_CHECK(hipFreeAsync(ptr, nullptr));
    CudaStreamManager::instance().onFree(byte_size);
  });
}

void fill_cuda(void* ptr, size_t size, Scalar t, DataType type) {
  LMP_DISPATCH_ALL_TYPES(type, [&]() {
    cudaVecFill(size, static_cast<scalar_t*>(ptr), static_cast<scalar_t>(t));
    LMP_CUDA_INTERNAL_ASSERT(hipGetLastError())
        << "fill_cuda: thrust::fill failed.";
  });
}

void resize_cuda(DataPtr dptr, size_t old_byte_size, size_t new_byte_size) {
  void* ptr = nullptr;
  LMP_CUDA_CHECK(hipMallocAsync(&ptr, new_byte_size, nullptr));
  LMP_CUDA_CHECK(hipMemcpyAsync(ptr, dptr.data(),
                                 std::min(old_byte_size, new_byte_size),
                                 hipMemcpyDeviceToDevice));

  auto* deleter = std::get_deleter<std::function<void(void*)>>(dptr.ptr);
  dptr = DataPtr(ptr, *deleter);
}

LMP_REGISTER_DISPATCH(ops::empty_stub, DeviceType::CUDA, empty_cuda);
LMP_REGISTER_DISPATCH(ops::fill_stub, DeviceType::CUDA, fill_cuda);
LMP_REGISTER_DISPATCH(ops::resize_stub, DeviceType::CUDA, resize_cuda);

void vecCopyHostToDevice(const void* src, void* dest, size_t size,
                         DataType src_dtype, DataType dest_dtype) {
  LMP_DISPATCH_ALL_TYPES(src_dtype, [&] {
    using src_type = scalar_t;
    LMP_DISPATCH_ALL_TYPES(dest_dtype, [&] {
      using dest_type = scalar_t;

      void* tmp = nullptr;
      LMP_CUDA_CHECK(hipMallocAsync(&tmp, size * sizeof(src_type), nullptr))
          << "copy_cpu to CUDA: hipMalloc for tmp failed.";
      LMP_CUDA_CHECK(hipMemcpyAsync(tmp, src, size * sizeof(src_type),
                                     hipMemcpyHostToDevice))
          << "copy_cpu to CUDA: hipMemcpy HtoD for tmp failed.";

      cudaVecCopy<src_type, dest_type>(size, static_cast<const src_type*>(tmp),
                                       static_cast<dest_type*>(dest));

      LMP_CUDA_INTERNAL_ASSERT(hipGetLastError())
          << "copy_cpu to CUDA: vecCopy kernel failed.";
      LMP_CUDA_CHECK(hipFreeAsync(tmp, nullptr))
          << "copy_cpu to CUDA: hipFreeAsync for tmp failed.";
      CudaStreamManager::instance().onFree(size * sizeof(src_type));
    });
  });
}

void copy_cuda(DeviceType to_device, const void* src, void* dest, size_t size,
               DataType src_dtype, DataType dest_dtype) {
  switch (to_device) {
    case DeviceType::CPU: {
      LMP_DISPATCH_ALL_TYPES(src_dtype, [&] {
        using src_type = scalar_t;
        LMP_DISPATCH_ALL_TYPES(dest_dtype, [&] {
          using dest_type = scalar_t;

          void* tmp = nullptr;
          LMP_CUDA_CHECK(
              hipMallocAsync(&tmp, size * sizeof(dest_type), nullptr))
              << "copy_cuda to CPU: hipMalloc for tmp failed.";

          cudaVecCopy<src_type, dest_type>(size,
                                           static_cast<const src_type*>(src),
                                           static_cast<dest_type*>(tmp));
          LMP_CUDA_INTERNAL_ASSERT(hipGetLastError())
              << "copy_cuda to CPU: vecCopy kernel failed.";
          LMP_CUDA_CHECK(hipMemcpyAsync(dest, tmp, size * sizeof(dest_type),
                                         hipMemcpyDeviceToHost))
              << "copy_cuda to CPU: hipMemcpy DtoH failed.";
          LMP_CUDA_CHECK(hipFreeAsync(tmp, nullptr))
              << "copy_cuda to CPU: hipFreeAsync for tmp failed.";
          CudaStreamManager::instance().onFree(size * sizeof(dest_type));
        });
      });
      break;
    }
    case DeviceType::CUDA: {
      LMP_DISPATCH_ALL_TYPES(src_dtype, [&] {
        using src_type = scalar_t;
        LMP_DISPATCH_ALL_TYPES(dest_dtype, [&] {
          using dest_type = scalar_t;

          void* tmp = nullptr;
          LMP_CUDA_CHECK(
              hipMallocAsync(&tmp, size * sizeof(dest_type), nullptr))
              << "copy_cuda to CUDA: hipMalloc for tmp failed.";

          cudaVecCopy<src_type, dest_type>(size,
                                           static_cast<const src_type*>(src),
                                           static_cast<dest_type*>(tmp));

          LMP_CUDA_INTERNAL_ASSERT(hipGetLastError())
              << "copy_cuda to CUDA: vecCopy kernel failed.";
          LMP_CUDA_CHECK(hipMemcpyAsync(dest, tmp, size * sizeof(dest_type),
                                         hipMemcpyDeviceToDevice))
              << "copy_cuda to CUDA: hipMemcpy DtoD failed.";
          LMP_CUDA_CHECK(hipFreeAsync(tmp, nullptr))
              << "copy_cuda to CUDA: hipFreeAsync for tmp failed.";
          CudaStreamManager::instance().onFree(size * sizeof(dest_type));
        });
      });
      break;
    }
    case DeviceType::Count:
      LMP_INTERNAL_ASSERT(false) << "DeviceType::Count is an internal utility.";
      break;
  }
}

template <typename U, typename V>
__global__ void cudaVecCopyKernel(size_t size, const U* in, V* out) {
  for (size_t i = (blockIdx.x * blockDim.x) + threadIdx.x; i < size;
       i += gridDim.x * blockDim.x) {
    out[i] = static_cast<V>(in[i]);
  }
}

template <typename U, typename V>
void cudaVecCopy(size_t size, const U* in, V* out) {
  size_t threads = 256;
  size_t blocks = std::min((size + threads - 1) / threads, 1024UL);
  cudaVecCopyKernel<U, V><<<blocks, threads>>>(size, in, out);
}

template <typename T>
__global__ void cudaVecFillKernel(size_t size, T* out, T value) {
  for (size_t i = (blockIdx.x * blockDim.x) + threadIdx.x; i < size;
       i += gridDim.x * blockDim.x) {
    out[i] = value;
  }
}

template <typename T>
void cudaVecFill(size_t size, T* out, T value) {
  size_t threads = 256;
  size_t blocks = std::min((size + threads - 1) / threads, 1024UL);
  cudaVecFillKernel<T><<<blocks, threads>>>(size, out, value);
}

#include "lamppp/tensor/supported_types.hpp"

#define INSTANTIATE_COPY(arg1_type, arg2_type)                              \
  template void cudaVecCopy<arg1_type, arg2_type>(size_t, const arg1_type*, \
                                                  arg2_type*);
#define INSTANTIATE_FILL(arg1_type) \
  template void cudaVecFill<arg1_type>(size_t, arg1_type*, arg1_type);

LMP_FOR_EACH_CARTESIAN_PRODUCT(INSTANTIATE_COPY, LMP_LIST_TYPES, LMP_LIST_TYPES)
LMP_FOR_EACH_CARTESIAN_PRODUCT(INSTANTIATE_FILL, LMP_LIST_TYPES)

#undef INSTANTIATE_COPY
#undef INSTANTIATE_FILL

LMP_REGISTER_DISPATCH(ops::copy_stub, DeviceType::CUDA, copy_cuda);

}  // namespace lmp::tensor::detail::cuda