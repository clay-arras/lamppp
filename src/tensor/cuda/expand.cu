#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <cuda/std/array>
#include <cuda/std/tuple>
#include "lamppp/tensor/cuda/expand.cuh"
#include "lamppp/tensor/cuda/list_ptr.cuh"
#include "lamppp/tensor/cuda/kernels.cuh"

namespace lmp::tensor::detail::cuda {

template <typename PtrList, typename OpFn>
__global__ void vectorized_expand_kernel(PtrList ptr_, OpFn fn_, size_t size,
                                         const CUDAOffsetUtil<kNArgs>* align) {
  size_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (i < size) {
    ::cuda::std::array offsets = align->get(i);
    ptr_.set_Out(i,
                 fn_(::cuda::std::get<1>(ptr_.fns)(ptr_.data[1], offsets[1]),
                     ::cuda::std::get<2>(ptr_.fns)(ptr_.data[2], offsets[2])));
  }
}

template <typename PtrList, typename OpFn>
void expand_kernel_launcher(PtrList ptr_, OpFn fn_, size_t size,
                            const CUDAOffsetUtil<kNArgs>* align) {
  size_t threads = 256;
  size_t blocks = (size + threads - 1) / threads;
  ListDevicePtr<CUDAOffsetUtil<kNArgs>> d_align(align, 1);
  vectorized_expand_kernel<<<blocks, threads>>>(ptr_, fn_, size, d_align.get());

  LMP_CUDA_INTERNAL_ASSERT(hipDeviceSynchronize())
      << "expand_kernel_launcher: kernel failed.";
}

template void expand_dispatch_handler<AddFunctor>(ExpandMetaHandler&);
template void expand_dispatch_handler<SubFunctor>(ExpandMetaHandler&);
template void expand_dispatch_handler<MulFunctor>(ExpandMetaHandler&);
template void expand_dispatch_handler<DivFunctor>(ExpandMetaHandler&);
template void expand_dispatch_handler<PowFunctor>(ExpandMetaHandler&);
template void expand_dispatch_handler<EqFunctor>(ExpandMetaHandler&);
template void expand_dispatch_handler<NeFunctor>(ExpandMetaHandler&);
template void expand_dispatch_handler<GeFunctor>(ExpandMetaHandler&);
template void expand_dispatch_handler<GtFunctor>(ExpandMetaHandler&);
template void expand_dispatch_handler<LeFunctor>(ExpandMetaHandler&);
template void expand_dispatch_handler<LtFunctor>(ExpandMetaHandler&);

}  // namespace lmp::tensor::detail::cuda